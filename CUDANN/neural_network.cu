#include "neural_network.hh"
#include "nn_exception.hh"

NeuralNetwork::NeuralNetwork(float learning_rate) :
	learning_rate(learning_rate)
{ }

NeuralNetwork::~NeuralNetwork() {
	for (auto layer : layers) {
		delete layer;
	}
}

void NeuralNetwork::addLayer(NNLayer* layer) {
	this->layers.push_back(layer);
}

Matrix NeuralNetwork::forward(Matrix X) {
	Matrix Z = X;
	// for every layer pass the output to the next layer
	for (auto layer : layers) {
		Z = layer->forward(Z);
	}

	Y = Z;
	return Y;
}

void NeuralNetwork::backprop(Matrix predictions, Matrix target) {
	// allocated memory if not already allocated
	dY.allocateMemoryIfNotAllocated(predictions.shape);
	// calculate cost
	Matrix error = bce_cost.dCost(predictions, target, dY);

	// reverse iteration to pass the error backwards from the last layer to the first
	for (auto it = this->layers.rbegin(); it != this->layers.rend(); it++) {
		error = (*it)->backprop(error, learning_rate);
	}
	// wait until all threads finish
	hipDeviceSynchronize();
}

std::vector<NNLayer*> NeuralNetwork::getLayers() const {
	return layers;
}