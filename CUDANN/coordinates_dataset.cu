#include "coordinates_dataset.hh"

CoodinatesDataset::CoodinatesDataset(size_t batch_size, size_t number_of_batches) :
	batch_size(batch_size), number_of_batches(number_of_batches)
{
	for (int i = 0; i < number_of_batches; i++) {
		batches.push_back(Matrix(Shape(batch_size, 2)));
		targets.push_back(Matrix(Shape(batch_size, 1)));

		batches[i].allocateMemory();
		targets[i].allocateMemory();

		for (int k = 0; k < batch_size; k++) {
			// load features
			batches[i][k] = static_cast<float>(rand()) / RAND_MAX - 0.5;
			batches[i][batches[i].shape.x + k] = static_cast<float>(rand()) / RAND_MAX - 0.5;
			
			// load targets
			if ((batches[i][k] > 0 && batches[i][batches[i].shape.x + k] > 0) || ((batches[i][k] < 0 && batches[i][batches[i].shape.x + k] < 0))) {
				targets[i][k] = 1;
			}
			else {
				targets[i][k] = 0;
			}
		}

		batches[i].copyHostToDevice();
		targets[i].copyHostToDevice();
	}
}

int CoodinatesDataset::getNumOfBatches() {
	return number_of_batches;
}

std::vector<Matrix>& CoodinatesDataset::getBatches() {
	return batches;
}

std::vector<Matrix>& CoodinatesDataset::getTargets() {
	return targets;
}